
#include <hip/hip_runtime.h>
// CUDA code adapted from: https://github.com/kctess5/range_libc/blob/deploy/includes/kernels.cu

#define DIST_THRESHOLD 0.0
#define STEP_COEFF 0.999

__device__ float distance(int x, int y, float *distMap, int width, int height) {
	return distMap[x * height + y];
}

__global__ void cuda_ray_marching(float * ins, float * outs, float * distMap, int width, int height, float max_range, int num_casts) {
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind >= num_casts) return;
	float x0 = ins[ind*3];
	float y0 = ins[ind*3+1];
	float theta = ins[ind*3+2];

	float ray_direction_x = cosf(theta);
	float ray_direction_y = sinf(theta);

	int px = 0;
	int py = 0;

	float t = 0.0;
	float out = max_range;
	// int iters = 0;
	while (t < max_range) {
		px = x0 + ray_direction_x * t;
		py = y0 + ray_direction_y * t;

		if (px >= width || px < 0 || py < 0 || py >= height) {
			out = max_range;
			break;
		}

		float d = distance(px,py, distMap, width, height);

		if (d <= DIST_THRESHOLD) {
			float xd = px - x0;
			float yd = py - y0;
			out =  sqrtf(xd*xd + yd*yd);
			break;
		}

		t += fmaxf(d * STEP_COEFF, 1.0);
		// iters ++;
	}
	outs[ind] = out;
}
